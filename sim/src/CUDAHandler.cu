#include "hip/hip_runtime.h"
#include "CUDAHandler.h"
#include "cudaKernels.cuh"
#include <hiprand/hiprand_kernel.h>
#include "cuda_utils.h"


__global__ void init_random(unsigned int seed, hiprandState_t* states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}


// 1D threads
__global__ void disturbeGameLife_kernel(GameLife* gameLife, float mousePosX, float mousePosY, int numberOfCells, float mouseRadius)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= numberOfCells) return;
    
    
    __shared__ float s_mousePosX;
    __shared__ float s_mousePosY;
    __shared__ float s_mouseRadiusSqr;
   
    

    if (threadIdx.x == 0) {
        s_mousePosX = mousePosX;
        s_mousePosY = mousePosY;
        s_mouseRadiusSqr = mouseRadius * mouseRadius;;
        
    }
    __syncthreads();

    // ! Yes! EARLY-EXIT STRATEGY  - Early AABB rejection to skip square root / dotProduct
    // vec2 pos = gameLife[i].position;
    // float dx = pos.x - s_mousePosX;
    // if(fabsf(dx) > mouseRadius) return;

    // float dy = pos.y - s_mousePosY;
    // if(fabsf(dy) > mouseRadius) return;
    
    // float distSq = dx * dx + dy * dy;

    // No EARLY-EXIT STRATEGY
    vec2 pos(s_mousePosX, s_mousePosY);
    float distSq = (gameLife[i].position - pos).magSq();

    if (distSq < s_mouseRadiusSqr) {
        gameLife[i].next ^= true;
        gameLife[i].color = make_uchar4(186, 186, 186, 255);
    }
}

__global__ void disturbeGameLife_kernel_2D(
    GameLife* gameLife,
    int gridRows, int gridCols,
    float cellSpacing,
    float mousePosX, float mousePosY,
    float mouseRadius)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r >= gridRows || c >= gridCols) return;

    int idx = r * gridCols + c;

    vec2 pos = gameLife[idx].position;

    // AABB rejection
    float dx = pos.x - mousePosX;
    if (fabsf(dx) > mouseRadius) return;

    float dy = pos.y - mousePosY;
    if (fabsf(dy) > mouseRadius) return;

    float distSq = dx * dx + dy * dy;
    if (distSq < mouseRadius * mouseRadius) {
        gameLife[idx].next ^= true;
    }
}

__global__ void disturbGameLife_kernel_windowed(
    GameLife* gameLife,
    int gridRows, int gridCols,
    float cellSpacing,
    float mouseX, float mouseY,
    float radius,
    int rowOffset, int colOffset)
{
    int localRow = blockIdx.y * blockDim.y + threadIdx.y;
    int localCol = blockIdx.x * blockDim.x + threadIdx.x;

    int globalRow = rowOffset + localRow;
    int globalCol = colOffset + localCol;

    if (globalRow >= gridRows || globalCol >= gridCols) return;

    int index = globalRow * gridCols + globalCol;

    vec2 pos = gameLife[index].position;

    float dx = pos.x - mouseX;
    float dy = pos.y - mouseY;

    if (fabsf(dx) > radius || fabsf(dy) > radius) return;

    float distSq = dx * dx + dy * dy;
    if (distSq < radius * radius) {
        gameLife[index].next ^= true;
    }
}

__global__ void disturbGameLife_kernel_windowed_shared(
    GameLife* gameLife,
    int gridRows, int gridCols,
    float cellSpacing,
    float mouseX, float mouseY,
    float radius,
    int rowOffset, int colOffset)
{
    int localRow = blockIdx.y * blockDim.y + threadIdx.y;
    int localCol = blockIdx.x * blockDim.x + threadIdx.x;

    int globalRow = rowOffset + localRow;
    int globalCol = colOffset + localCol;

    if (globalRow >= gridRows || globalCol >= gridCols) return;

    // --- Shared memory for read-only constants
    __shared__ float s_mouseX;
    __shared__ float s_mouseY;
    __shared__ float s_radiusSq;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        s_mouseX = mouseX;
        s_mouseY = mouseY;
        s_radiusSq = radius * radius;
    }

    __syncthreads();  // make sure all threads see the shared values

    int index = globalRow * gridCols + globalCol;

    vec2 pos = gameLife[index].position;

    float dx = pos.x - s_mouseX;
    if (fabsf(dx) > radius) return;

    float dy = pos.y - s_mouseY;
    if (fabsf(dy) > radius) return;

    float distSq = dx * dx + dy * dy;
    if (distSq < s_radiusSq) {
        gameLife[index].next ^= true;
    }
}





__global__ void drawParticles_kernel(hipSurfaceObject_t surface, GameLife* particles, int numberParticles, int width, int height, float zoom, float panX, float panY){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numberParticles || !particles[i].alive) return;
    GameLife gl = particles[i];
    vec2 pos = gl.position;
    float radius = gl.radius * zoom;
    int x0 = (int)(width / 2.0f + (pos.x + panX) * zoom);
    int y0 = (int)(height / 2.0f + (pos.y + panY) * zoom);
    
    drawFilledCircle(surface, x0, y0, radius, gl.color, width, height);
} 

__global__ void commitNextState_kernel(GameLife* gamelife, int totalParticles) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;

    gamelife[i].alive = gamelife[i].next;
    gamelife[i].next = false;
}

__global__ void activate_gameOfLife_kernel(GameLife* gamelife, int totalParticles, int gridRows, int gridCols) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;
  
    int row = i / gridCols;
    int col = i % gridCols;
    int aliveCount = 0;
    
    for (int dr = -1; dr <= 1; ++dr) {
        for (int dc = -1; dc <= 1; ++dc) {
            if (dr == 0 && dc == 0) continue; // skip self

            int nr = row + dr;
            int nc = col + dc;
            // * Check if neighbors are within the grid bounds
            if (nr >= 0 && nr < gridRows && nc >= 0 && nc < gridCols) {
                int j = nr * gridCols + nc;
                // * Check if neighbors are alive and count them
                if (gamelife[j].alive) aliveCount++;
            }
        }
    }
    
    // Apply rules
    if (gamelife[i].alive )
        gamelife[i].next = (aliveCount == 2 || aliveCount == 3); // stays alive or not
    else {
        gamelife[i].next = (aliveCount == 3);
    }
}

__device__ float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

__device__ float tanhMapped(float x) {
    return 0.5f * (tanhf(x) + 1.0f);
}

__device__ float reluClamped(float x) {
    return fminf(fmaxf(x, 0.0f), 1.0f);
}
__device__ float reluProb(float x) {
    // scale to a more usable range
    float scaled = 0.25f * x;  // adjust scale as needed
    return fminf(fmaxf(scaled, 0.0f), 1.0f);
}



__global__ void activate_gameOfLife_convolution_kernel(hiprandState_t* states, GameLife* gamelife, int totalParticles, int gridRows, int gridCols, GameMode gameMode, float threshold, float edgeWeight, float cornerWeight) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= totalParticles) return;

    int row = i / gridCols;
    int col = i % gridCols;

    float neighborSum = 0.0f;
    int aliveCount = 0;

    // Example kernel weights (symmetric, Gaussian-like)
    // const float kernel[3][3] = {
    //     {0.5f, 1.0f, 0.5f},
    //     {1.0f, 0.0f, 1.0f},  // center weight is ignored
    //     {0.5f, 1.0f, 0.5f}
    // };
    // Define your kernel weights dynamically
    const float kernel[3][3] = {
        {cornerWeight, edgeWeight, -cornerWeight},
        {edgeWeight,   0.0f,       -edgeWeight},
        {cornerWeight, -edgeWeight, -cornerWeight}
    };


    for (int dr = -1; dr <= 1; ++dr) {
        for (int dc = -1; dc <= 1; ++dc) {
            if (dr == 0 && dc == 0) continue;

            int nr = row + dr;
            int nc = col + dc;
            if (nr >= 0 && nr < gridRows && nc >= 0 && nc < gridCols) {
                int j = nr * gridCols + nc;
                neighborSum += (gamelife[j].alive ? 1.0f : 0.0f) * kernel[dr + 1][dc + 1];
                aliveCount += gamelife[j].alive ? 1 : 0;

            }
        }
    }

    if (gameMode == gameOfLife) {
        if (gamelife[i].alive )
            gamelife[i].next = (aliveCount == 2 || aliveCount == 3); // stays alive or not
        else {
            gamelife[i].next = (aliveCount == 3);
        }
    } else if (gameMode == sigmoidF) {
        // float threshold = 3.0f;  // similar to GoL but flexible
        float probability = sigmoid(neighborSum - threshold);
        // gamelife[i].next = (probability > 0.5f);   // no stochastic
        hiprandState_t x = states[i];
        gamelife[i].next = (hiprand_uniform(&x) <= probability); // stochastic : random() < probability
        states[i] = x; // save back

    } else if (gameMode == hyperbolicTanF) {
        float probability = tanhMapped(neighborSum - threshold);
        hiprandState_t x = states[i];
        gamelife[i].next = (hiprand_uniform(&x) <= probability);
        states[i] = x; // save back
        // gamelife[i].next = (probability > 0.5f);
    } else if (gameMode == reLuF) {
        float probability = reluProb(neighborSum - threshold);
        hiprandState_t x = states[i];
        gamelife[i].next = (hiprand_uniform(&x) <= probability);
        states[i] = x; // save back
    }

    // // float threshold = 3.0f;  // similar to GoL but flexible
    // float probability = sigmoid(neighborSum - threshold);
    // // gamelife[i].next = (probability > 0.5f);  // or experiment with stochastic: random() < probability
    // hiprandState_t x = states[i];
    // gamelife[i].next = (hiprand_uniform(&x) <= probability);
    // states[i] = x; // save back

    // float probability = sigmoid(neighborSum - threshold);
    // if (probability > 0.7f) gamelife[i].next = true;
    // else if (probability < 0.1f) gamelife[i].next = false;
    // else gamelife[i].next = gamelife[i].alive;  // keep previous state
}





CUDAHandler* CUDAHandler::instance = nullptr;

CUDAHandler::CUDAHandler(int width, int height, GLuint textureID) :  width(width), height(height)
{
    hipGraphicsGLRegisterImage(&cudaResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    instance = this; // store global reference (to be used for mouse and imGui User Interface (UI) operations)
    center = vec2(width / 2.0f, height / 2.0f);
    screenRatio = static_cast<float>(height) / width;
    
}

CUDAHandler::~CUDAHandler()
{
    hipFree(d_gameLife);

    hipGraphicsUnregisterResource(cudaResource);
    
}
// _________________________________________________________________________//
void CUDAHandler::updateDraw(float dt)
{
    this->dt = dt;
    framesCount++;


    static Settings previousSettings;
    Settings currentSettings = {
        // .gameMode = gameMode,
        .numberOfParticles = numberOfParticles,
        .particleRadius = particleRadius,
        .restLength = restLength,
        .option = option,
        .widthFactor = widthFactor,
        .gridSize = gridSize,
        .thickness = thickness,
        .ringSpacing = ringSpacing,
        .spacing = spacing,
        .band = band,
        .blockSize = blockSize,
        .diagonalBand = diagonalBand,
        .border = border,
        .rule = rule
    };
    

    if (gamelife.empty() || currentSettings != previousSettings) {
        framesCount = 0;
        initGameLife();
        previousSettings = currentSettings;
    }
    
    
    // GameLife* d_gameLife;
    // checkCuda(hipMalloc(&d_gameLife, gamelife.size() * sizeof(GameLife)));
    // checkCuda(hipMemcpy(d_gameLife, gamelife.data(), gamelife.size() * sizeof(GameLife), hipMemcpyHostToDevice));
    
    
    if(startSimulation) {
        activateGameLife(d_gameLife);
    }
    // checkCuda(hipMemcpy(gamelife.data(), d_gameLife, gamelife.size() * sizeof(GameLife), hipMemcpyDeviceToHost));
    

    hipSurfaceObject_t surface = MapSurfaceResouse(); 
   
    clearGraphicsDisply(surface, DARK);

    // draw samples to check ZOOM & PAN
    
    // drawCircle_kernel<<<1, 1>>>(surface, width, height, center.x, center.y, 200, SUN_YELLOW, 1, 4, zoom, panX, panY);
    // drawGlowingCircle_kernel<<<1, 1>>>(surface, width, height, center.x, center.y, 500, RED_MERCURY, 1.5f, zoom, panX, panY);
    // drawRing(surface, center, 500, 4, BLUE_PLANET);

    // drawGlowingCircle(surface, center, 500, 1.5, RED_MERCURY );

    

    drawGameLife(surface, d_gameLife);

    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());

    // hipFree(d_gameLife);

    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &cudaResource);
}

//________________________________________________________________________//

void CUDAHandler::clearGraphicsDisply(hipSurfaceObject_t &surface, uchar4 color)
{
    int threads = 16; 
    dim3 clearBlock(threads, threads);
    dim3 clearGrid((width + clearBlock.x -1) / clearBlock.x, (height + clearBlock.y - 1) / clearBlock.y);
    clearSurface_kernel<<<clearGrid, clearBlock>>>(surface, width, height, color);
}

void CUDAHandler::drawGlowingCircle(hipSurfaceObject_t &surface, vec2 position, float radius, float glowExtent, uchar4 color)
{
    // Map world center to screen center
    float screen_cx = (position.x + panX) * zoom + width / 2.0f;
    float screen_cy = (position.y + panY) * zoom + height / 2.0f;

    // Calculate radius in screen pixels
    float screen_radius = radius * zoom;
    float screen_glowRadius = glowExtent * screen_radius;

    int xmin = max(0, (int)(screen_cx - screen_glowRadius));
    int xmax = min(width-1, (int)(screen_cx + screen_glowRadius));
    int ymin = max(0, (int)(screen_cy - screen_glowRadius));
    int ymax = min(height-1, (int)(screen_cy + screen_glowRadius));


    
    // // Calculate bounding box   // if not zoom , nor panx, nor pany involved
    // float glowRadius = glowExtent * radius;
    // int xMin = max(0, (int)(position.x - glowRadius));
    // int xMax = min(width - 1, (int)(position.x + glowRadius));
    // int yMin = max(0, (int)(position.y - glowRadius));
    // int yMax = min(height - 1, (int)(position.y + glowRadius));

    int drawWidth   = xmax - xmin + 1;
    int drawHeight  = ymax - ymin + 1;

    dim3 blockSize(16, 16);
    dim3 gridSize ((drawWidth + blockSize.x - 1) / blockSize.x, (drawHeight + blockSize.y -1) / blockSize.y);
    drawGlowingCircle_kernel<<<gridSize, blockSize>>>(surface, width, height, position.x, position.y, radius,  color, 1.5f, xmin, ymin, zoom, panX, panY);
}

void CUDAHandler::drawRing(hipSurfaceObject_t &surface, vec2 position, float radius, float thickness, uchar4 color)
{
    // Map world center to screen center
    float screen_cx = (position.x + panX) * zoom + width / 2.0f;
    float screen_cy = (position.y + panY) * zoom + height / 2.0f;

    // Calculate radius in screen pixels
    float screen_radius = radius * zoom;

    int xmin = max(0, (int)(screen_cx - screen_radius - thickness));
    int xmax = min(width - 1, (int)(screen_cx + screen_radius + thickness));
    int ymin = max(0, (int)(screen_cy - screen_radius - thickness));
    int ymax = min(height - 1, (int)(screen_cy + screen_radius + thickness));

    int drawWidth   = xmax - xmin + 1;
    int drawHeight  = ymax - ymin + 1;

    dim3 blockSize(16, 16);
    dim3 gridSize ((drawWidth + blockSize.x - 1) / blockSize.x, (drawHeight + blockSize.y -1) / blockSize.y);
    // Pass world-space center (not screen-space) to the kerne
    // drawRing_kernel<<<gridSize, blockSize>>>(surface, width, height, position.x, position.y, radius,  color, thickness, xmin, ymin, zoom, panX, panY);
    drawRing_sharedMemory_kernel<<<gridSize, blockSize>>>(surface, width, height, position.x, position.y, radius,  color, thickness, xmin, ymin, zoom, panX, panY);



}

void CUDAHandler::activateGameLife()
{
    
    for (auto &gl : gamelife) {
        gl.alive = gl.next;   // next generation is the current generation
        gl.next = false;

    }
    
    int aliveCount;
    // 1. For every particle, calculate its row and column
    for (int i = 0; i < gamelife.size(); ++i) {
        int row = i / gridCols;
        int col = i % gridCols;
        aliveCount = 0;        
         // 2. Loop over all 8 neighbors (including diagonals)
        for (int dr = -1; dr <= 1; ++dr) {
            for (int dc = -1; dc <= 1; ++dc) {
                if (dr == 0 && dc == 0) continue; // skip self 
               
                int nr = row + dr;
                int nc = col + dc;
                
                // 3.  Check if neighbor is within the grid bounds
                if (nr >= 0 && nr < gridRows && nc >= 0 && nc < gridCols) {
                    int j = nr * gridCols + nc;
                    // 4. check if the neighbors are alive
                    if (gamelife[j].alive) aliveCount++;
                }
            }
        }
        gamelife[i].aliveNeighbors = aliveCount;  
    }
    


    for (auto &gl : gamelife) {
        if (!gl.alive && gl.aliveNeighbors == 3) gl.next = true;  // revives : reproduction
        if (gl.alive && gl.aliveNeighbors < 2)   gl.next = false; // dies : underpopulation
        if (gl.alive && gl.aliveNeighbors > 3)   gl.next = false;  // dies : overpopulation
        if (gl.alive && (gl.aliveNeighbors == 2 || gl.aliveNeighbors == 3)) gl.next = true;  // stays alive

    }
}

void CUDAHandler::activateGameLife(GameLife* &d_gameLife)
{
    int threads = 256;
    int blocks = (gamelife.size() + threads - 1) / threads;
    commitNextState_kernel<<<blocks, threads>>> (d_gameLife, gamelife.size());
    checkCuda(hipDeviceSynchronize());

    //generate random seed to be used in rayTracer kernel
    int num_threads = threads * blocks;
    hiprandState_t* d_states;
    checkCuda(hipMalloc(&d_states, num_threads * sizeof(hiprandState_t)));
    init_random<<<blocks, threads>>>(time(0), d_states);
    checkCuda(hipDeviceSynchronize() );

    activate_gameOfLife_convolution_kernel<<<blocks, threads>>>(d_states, d_gameLife, gamelife.size(), gridRows, gridCols, gameMode, sigmoidThreshold, kernelWeightEdge, kernelWeightCorner);
    // activate_gameOfLife_kernel<<<blocks, threads>>>(d_gameLife, gamelife.size(), gridRows, gridCols);
    hipFree(d_states);
}

void CUDAHandler::initGameLife()
{
    
    gamelife.clear();
    startSimulation = false;
    setGroupOfParticles(numberOfParticles, {16, 9});
    // setGroupOfParticles({16, 9});
    checkCuda(hipMalloc(&d_gameLife, gamelife.size() * sizeof(GameLife)));
    checkCuda(hipMemcpy(d_gameLife, gamelife.data(), gamelife.size() * sizeof(GameLife), hipMemcpyHostToDevice));

}

int2 CUDAHandler::calculateGrid(int n, int a, int b)
{
    double targetRatio = static_cast<double>(a) / b;
    double bestDiff = std::numeric_limits<double>::max();
    int bestRows = 1, bestCols = n;

    for (int rows = 1; rows <= n; ++rows) {
        int cols = (n + rows - 1) / rows; // ceil(n / rows)
        double currentRatio = static_cast<double>(cols) / rows;
        double diff = std::abs(currentRatio - targetRatio);

        if (diff < bestDiff) {
            bestDiff = diff;
            bestRows = rows;
            bestCols = cols;
        }
    }

    return {bestRows, bestCols};
}

int2 CUDAHandler::calculateGridWithRatio(float ratioX, float ratioY)
{
    float cellSize = 2.0f * particleRadius + restLength;

    int maxCols = static_cast<int>(width  / cellSize);
    int maxRows = static_cast<int>(height / cellSize);

    float aspect = 1.0 ; //ratioX / ratioY;

    // Fit the grid while maintaining the ratio and staying within bounds
    int cols = std::min(maxCols, static_cast<int>(maxRows * aspect));
    int rows = std::min(maxRows, static_cast<int>(cols / aspect));

    return int2{rows, cols};
}

int2 CUDAHandler::calculateGridClamped(int n, int a, int b)
{
    double targetRatio = static_cast<double>(a) / b;
    double bestDiff = std::numeric_limits<double>::max();

    float cellSize = 2.0f * particleRadius + restLength;

    int maxCols = static_cast<int>(width  / cellSize);
    int maxRows = static_cast<int>(height / cellSize);

    int bestRows = 1, bestCols = n;

    for (int rows = 1; rows <= maxRows; ++rows) {
        int cols = (n + rows - 1) / rows; // ceil(n / rows)
        if (cols > maxCols) continue;     // skip: doesn't fit on screen

        double currentRatio = static_cast<double>(cols) / rows;
        double diff = std::abs(currentRatio - targetRatio);

        if (diff < bestDiff) {
            bestDiff = diff;
            bestRows = rows;
            bestCols = cols;
        }
    }

    return {bestRows, bestCols};
}

void CUDAHandler::drawGameLife(hipSurfaceObject_t &surface, GameLife *&d_gameLife)
{
    int threads = 256;
    int blocks = (gamelife.size() + threads -1 ) / threads;
    drawParticles_kernel<<<blocks, threads>>>(surface, d_gameLife, gamelife.size(), width, height, zoom, panX, panY);
}

void CUDAHandler::disturbeGameLife(vec2 mousePosition)
{
    // for (int i = 0; i < gamelife.size(); ++i) {

    //     float d2 = (gamelife[i].position - mousePosition).magSq();
    //     if (d2 <  mouseCursorRadius * mouseCursorRadius) {
            
    //         gamelife[i].next ^= true;

    //     }

    // }

    // 1D kernel //
    // checkCuda(hipMemcpy(d_gameLife, gamelife.data(), gamelife.size() * sizeof(GameLife), hipMemcpyHostToDevice));
    int threads = 256;
    int blocks = (gamelife.size() + threads - 1) / threads;

    disturbeGameLife_kernel<<<blocks, threads>>>(d_gameLife, mousePosition.x, mousePosition.y, gamelife.size(), mouseCursorRadius);

    // checkCuda(hipMemcpy(gamelife.data(), d_gameLife, gamelife.size() * sizeof(GameLife), hipMemcpyDeviceToHost));


    // 2D Kernel
    // // checkCuda(hipMemcpy(d_gameLife, gamelife.data(), gamelife.size() * sizeof(GameLife), hipMemcpyHostToDevice));
    // dim3 blockSize(16, 16);
    // dim3 gridSize((gridCols + blockSize.x - 1) / blockSize.x, (gridRows + blockSize.y - 1) / blockSize.y);

    // disturbeGameLife_kernel_2D<<<gridSize, blockSize>>>(d_gameLife, gridRows, gridCols, restLength, mousePosition.x, mousePosition.y, mouseCursorRadius);
    
    // // checkCuda(hipMemcpy(gamelife.data(), d_gameLife, gamelife.size() * sizeof(GameLife), hipMemcpyDeviceToHost));

   
    


    // Compute min/max row/col range on host
    // checkCuda(hipMemcpy(d_gameLife, gamelife.data(), gamelife.size() * sizeof(GameLife), hipMemcpyHostToDevice));
    // int minCol = max(0, int((mousePosition.x - mouseCursorRadius - topLeft.x) / restLength));
    // int maxCol = min(gridCols, int((mousePosition.x + mouseCursorRadius - topLeft.x) / restLength));
    // int minRow = max(0, int((mousePosition.y - mouseCursorRadius - topLeft.y) / restLength));
    // int maxRow = min(gridRows, int((mousePosition.y + mouseCursorRadius - topLeft.y) / restLength));
    // int drawWidth   = maxCol - minCol + 1;
    // int drawHeight  = maxRow - minRow + 1;
    
    // dim3 blockSize(16, 16);
    // dim3 gridSize((drawWidth + blockSize.x - 1) / blockSize.x, (drawHeight + blockSize.y - 1) / blockSize.y);
    // // disturbGameLife_kernel_windowed<<<gridSize, blockSize>>>(d_gameLife, gridRows, gridCols, restLength, mousePosition.x, mousePosition.y, mouseCursorRadius, minRow, minCol);
    // disturbGameLife_kernel_windowed_shared<<<gridSize, blockSize>>>(d_gameLife, gridRows, gridCols, restLength, mousePosition.x, mousePosition.y, mouseCursorRadius, minRow, minCol);
    // // checkCuda(hipMemcpy(gamelife.data(), d_gameLife, gamelife.size() * sizeof(GameLife), hipMemcpyDeviceToHost));




}

void CUDAHandler::setGroupOfParticles(int totalParticles, int2 ratio, bool anchors )
{
    
    // ratio refers to the proportion of length vs width
    int2 grid = calculateGrid(totalParticles, ratio.x,ratio.y);
    // restLength = 2 * particleRadius;
    // int2 grid = calculateGridClamped(totalParticles, ratio.x,ratio.y);
    int rows = grid.x;
    int cols = grid.y;

    // printf("Rows: %d  -  Cols: %d - total: %d\n", rows, cols, rows * cols);

    gridRows = rows;
    gridCols = cols;    

    // int offset = width / 2.0f - (cols - 1) * particleRadius;    
    // float offset = width / 2.0f - (cols - 1) * restLength / 2.0f;
    float offsetX = (width  - (cols - 1) * restLength) / 2.0f;
    float offsetY = (height - (rows - 1) * restLength) / 2.0f;
    topLeft = vec2(offsetX, offsetY);


    // topLeft = vec2(offset, top);
    
    int rowsSize = widthFactor * gridRows;
    int colsSize = widthFactor * gridCols * screenRatio;  // screen ratio for correctness

    // Place particles in a 2D grid at restLength spacing
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            float x = topLeft.x + c * restLength;
            float y = topLeft.y + r * restLength;
            GameLife gl;
            gl.position = vec2(x,y);
            gl.radius = particleRadius;
            switch(option){
                case 0:   // grid
                    
                    if (c % gridSize == 0 || r % gridSize == 0) {
                        gl.alive = gl.next = true;
                        gl.color = WHITE;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GREEN;
                    }

                    
                    break;
                case 1: // Vertical
                    if ((c / colsSize) % 2 == 0) {
                        gl.alive = gl.next = true;      // cell is ON
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;     // cell is OFF
                        gl.color = GOLD;
                    }
                    break;
                case 2: // horizontal
                    if ((r / rowsSize) % 2 == 0) {
                        gl.alive = gl.next = true;      // cell is ON  
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;     // cell is OFF
                        gl.color = GOLD;
                    }
                    break;
                case 3:    // checkered
                    
                    if ((r / rowsSize) % 2 == 0 && c / colsSize % 2 == 0) { 
                        gl.alive = gl.next = true;      // cell is ON
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;     // cell is OFF
                        gl.color = GOLD;
                    }
                    break;
                case 4: { // diagonal
                    int band = 0;
                    if (abs(r - c) < band ) {
                        gl.alive = gl.next = true;
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GOLD;
                    }
                    break;
                }
                case 5: {  // x shape
                        int band = 0;
                        int centerOffset = cols - rows;
                        if (abs((r) - (c - centerOffset / 2)) <= band || abs((r) + (c - centerOffset / 2) - (rows - 1)) <= band) {
                        // if (r == c || r + c == rows - 1) {
                        // if (abs(r - c) <= band || abs(r + c - (rows - 1)) <= band) {

                        gl.alive = gl.next = true;
                        gl.color = RED_MERCURY;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GOLD;
                    }
                    break;
                }
                case 6: { // Circle
                    float centerX = cols / 2.0f;
                    float centerY = rows / 2.0f;
                    int gap = 150;
                    int gapSq = gap * gap;
                    float radiusSquared = (rows / 3.0f) * (rows / 3.0f);
                    float radiusSquared2 = (rows / 6.0f) * (rows / 6.0f);
                    float radiusSquared3 = (rows / 9.0f) * (rows / 9.0f);
                    float dx = c - centerX;
                    float dy = r - centerY;
                    float dist = dx * dx + dy * dy;
                    if (dist <= radiusSquared && dist > radiusSquared2 + gapSq) {
                        gl.alive = gl.next = true;
                        gl.color = BLUE_PLANET;
                    } else if (dist > radiusSquared2 && dist < radiusSquared3 + gapSq ){
                        gl.alive = gl.next = true;
                        gl.color = SUN_YELLOW;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = URANUS_BLUE;
                    }
                    break;
                    }
                case 7: {
                    float cx = cols / 2.0f;
                    float cy = rows / 2.0f;
                
                    float dx = c - cx;
                    float dy = r - cy;
                    float dist = sqrtf(dx * dx + dy * dy);
                    float angle = atan2f(dy, dx);  // [-π, π]
                    angle = angle < 0 ? angle + 2.0f * M_PI : angle;
                
                    // Parameters
                    // float spacing = 6.0f;        // radial spacing per full rotation (~coil tightness)
                    // float thickness = 2.5f;      // thickness of the spiral band
                
                    // Spiral formula: r = spacing * theta
                    float r_cw = angle * spacing;
                    float diff_cw = fabs(dist - r_cw);
                
                    // Opposite spiral
                    float r_ccw = (2.0f * M_PI - angle) * spacing;
                    float diff_ccw = fabs(dist - r_ccw);
                
                    if (diff_cw < thickness || diff_ccw < thickness) {
                        gl.alive = gl.next = true;
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = PINK;
                    }
                    break;
                }
                case 8: { // border
                    int border = 50;  // thickness of border
                    if (r < border || r >= rows - border || c < border || c >= cols - border) {
                        gl.alive = gl.next = true;
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GOLD;
                    }
                    break;
                }
                case 9: {  // double border
                    int outer = 1;  // outer thickness
                    int inner = 50;  // inner offset
                    bool isOuter = (r < outer || r >= rows - outer || c < outer || c >= cols - outer);
                    bool isInner = (r >= inner && r < rows - inner && c >= inner && c < cols - inner);
                    if (isOuter || isInner) {
                        gl.alive = gl.next = true;
                        gl.color = NEPTUNE_PURPLE;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = SUN_YELLOW;
                    }
                    break;
                }
                case 10: { // concentric Rings
                    float cx = cols / 2.0f;
                    float cy = rows / 2.0f;
                    float dx = c - cx;
                    float dy = r - cy;
                    float dist = sqrtf(dx * dx + dy * dy);
                
                    //* ringSpacing :controls distance between rings
                    //* thickness : ring band thickness
                
                    float modVal = fmodf(dist, ringSpacing);
                    if (modVal < thickness) {
                        gl.alive = gl.next = true;
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = NEPTUNE_PURPLE;
                    }
                    break;
                }
                case 11: { // Radial beam
                    float cx = cols / 2.0f;
                    float cy = rows / 2.0f;
                    float dx = c - cx;
                    float dy = r - cy;
                
                    float angle = atan2f(dy, dx);  // range: [-π, π]
                    angle = angle < 0 ? angle + 2.0f * M_PI : angle;  // normalize to [0, 2π]
                
                    int numBeams = 16;         // number of sun rays
                    float beamWidth = M_PI * 2.0f / numBeams;  // angle between beams
                
                    int beamIndex = (int)(angle / beamWidth);
                    if (beamIndex % 2 == 0) {
                        gl.alive = gl.next = true;
                        gl.color = SUN_YELLOW;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GREEN;
                    }
                    break;
                }
                case 12: {  // Animated Rotating Sunbeam
                    float cx = cols / 2.0f;
                    float cy = rows / 2.0f;
                    float dx = c - cx;
                    float dy = r - cy;
                
                    float angle = atan2f(dy, dx);
                    angle = angle < 0 ? angle + 2.0f * M_PI : angle;
                
                    int numBeams = 16;
                    float beamWidth = 2.0f * M_PI / numBeams;
                
                    float angularOffset = fmodf(framesCount * dt * 0.5f, 2.0f * M_PI);  // rotate over time
                    angle += angularOffset;
                
                    int beamIndex = (int)(angle / beamWidth);
                    if (beamIndex % 2 == 0) {
                        gl.alive = gl.next = true;
                        gl.color = SUN_YELLOW;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = URANUS_BLUE;
                    }
                    break;
                }
            case 13: {
                // int blockSize = 6;      // size of each square block
                // int band = 1;           // diagonal thickness
            
                int blockRow = r / blockSize;
                int blockCol = c / blockSize;
            
                int localR = r % blockSize;
                int localC = c % blockSize;
            
                // Diagonal type: choose one or alternate
                bool useForwardSlash = true;  // true = '/', false = '\'
            
                // Optional: alternate slashes like a checker
                // if ((blockRow + blockCol) % 2 == 0) useForwardSlash = true;
                // else useForwardSlash = false;
            
                bool isDiagonal = false;
            
                if (useForwardSlash) {
                    isDiagonal = abs(localR + localC - (blockSize - 1)) <= band;
                } else {
                    isDiagonal = abs(localR - localC) <= band;
                }
            
                if (isDiagonal) {
                    gl.alive = gl.next = true;
                    gl.color = ORANGE;
                } else {
                    gl.alive = gl.next = false;
                    gl.color = TAN;
                }
                break;
            }
        case 14: {
            // int blockSize = 6;  // size of each square
            // int border = 1;     // thickness of grid lines
            // int diagonalBand = 1;  // diagonal thickness
        
            int blockRow = r / blockSize;
            int blockCol = c / blockSize;
        
            int localR = r % blockSize;
            int localC = c % blockSize;
        
            bool isBorder = (localR < border || localR >= blockSize - border ||
                                localC < border || localC >= blockSize - border);
        
            // Diagonal type: '/' or '\' or alternating
            bool useForwardSlash = ((blockRow + blockCol) % 2 == 0);  // alternate per tile
        
            bool isDiagonal = false;
            if (useForwardSlash) {
                isDiagonal = abs(localR + localC - (blockSize - 1)) <= diagonalBand;
            } else {
                isDiagonal = abs(localR - localC) <= diagonalBand;
            }
        
            if (isBorder || isDiagonal) {
                gl.alive = gl.next = true;
                gl.color = make_uchar4(255, 200, 50, 255);  // gold-orange
            } else {
                gl.alive = gl.next = false;
                gl.color = make_uchar4(20, 20, 20, 255);  // dark background
            }
            break;
        }
        // case 15: { // rule 30
        //     int center = cols / 2;
        //     if (r == 0) {
        //         // First row: set single center cell alive
        //         gl.alive = gl.next = (c == center);
        //     } else {
        //         // Read previous row
        //         int rowAbove = r - 1;
        //         int idxLeft = rowAbove * cols + c - 1;
        //         int idxCenter = rowAbove * cols + c;
        //         int idxRight = rowAbove * cols + c + 1;
        
        //         bool left = (c > 0)              ? gamelife[idxLeft].alive   : false;
        //         bool mid  =                        gamelife[idxCenter].alive;
        //         bool right= (c < cols - 1)       ? gamelife[idxRight].alive  : false;
        
        //         // Apply Rule 30 logic:  new = left XOR (mid OR right)
        //         gl.alive = gl.next = left ^ (mid || right);
        //     }
        
        //     gl.color = gl.alive ? make_uchar4(255, 255, 255, 255) : make_uchar4(0, 0, 0, 255);
        //     break;
        // }
        case 15: {
            int center = cols / 2;
        
            // Expose rule via ImGui slider (external to this loop)
            // extern uint8_t rule;
        
            if (r == 0) {
                // Seed: single dot at center
                gl.alive = gl.next = (c == center);
            } else {
                int rowAbove = r - 1;
                int idxLeft = rowAbove * cols + c - 1;
                int idxMid  = rowAbove * cols + c;
                int idxRight= rowAbove * cols + c + 1;
        
                bool left  = (c > 0)         ? gamelife[idxLeft].alive : false;
                bool mid   = gamelife[idxMid].alive;
                bool right = (c < cols - 1)  ? gamelife[idxRight].alive : false;
        
                int pattern = (left << 2) | (mid << 1) | right;
                gl.alive = gl.next = ((rule >> pattern) & 1);
            }
        
            gl.color = gl.alive ? WHITE : RED_MERCURY;
            break;
        }
                
                default: 
                    break;

            }
            
            gamelife.push_back(gl);
        }
    }
}

void CUDAHandler::setGroupOfParticles(int2 ratio)
{
    // ratio refers to the proportion of length vs width
    restLength = 2 * particleRadius;
    int2 grid = calculateGridWithRatio(ratio.x, ratio.y);
    int rows = grid.x;
    int cols = grid.y;

    // printf("Rows: %d  -  Cols: %d - total: %d\n", rows, cols, rows * cols);

    gridRows = rows;
    gridCols = cols;    

    // int offset = width / 2.0f - (cols - 1) * particleRadius;    
    // float offset = width / 2.0f - (cols - 1) * restLength / 2.0f;
    float offsetX = (width  - (cols - 1) * restLength) / 2.0f;
    float offsetY = (height - (rows - 1) * restLength) / 2.0f;
    topLeft = vec2(offsetX, offsetY);


    // topLeft = vec2(offset, top);
    
    int rowsSize = widthFactor * gridRows;
    int colsSize = widthFactor * gridCols * screenRatio;  // screen ratio for correctness

    // Place particles in a 2D grid at restLength spacing
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            float x = topLeft.x + c * restLength;
            float y = topLeft.y + r * restLength;
            GameLife gl;
            gl.position = vec2(x,y);
            gl.radius = particleRadius;
            switch(option){
                case 0:   // grid
                    
                    if (c % gridSize == 0 || r % gridSize == 0) {
                        gl.alive = gl.next = true;
                        gl.color = WHITE;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GREEN;
                    }

                    
                    break;
                case 1: // Vertical
                    if ((c / colsSize) % 2 == 0) {
                        gl.alive = gl.next = true;      // cell is ON
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;     // cell is OFF
                        gl.color = GOLD;
                    }
                    break;
                case 2: // horizontal
                    if ((r / rowsSize) % 2 == 0) {
                        gl.alive = gl.next = true;      // cell is ON  
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;     // cell is OFF
                        gl.color = GOLD;
                    }
                    break;
                case 3:    // checkered
                    
                    if ((r / rowsSize) % 2 == 0 && c / colsSize % 2 == 0) { 
                        gl.alive = gl.next = true;      // cell is ON
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;     // cell is OFF
                        gl.color = GOLD;
                    }
                    break;
                case 4: { // diagonal
                    int band = 0;
                    if (abs(r - c) < band ) {
                        gl.alive = gl.next = true;
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GOLD;
                    }
                    break;
                }
                case 5: {  // x shape
                        int band = 0;
                        int centerOffset = cols - rows;
                        if (abs((r) - (c - centerOffset / 2)) <= band || abs((r) + (c - centerOffset / 2) - (rows - 1)) <= band) {
                        // if (r == c || r + c == rows - 1) {
                        // if (abs(r - c) <= band || abs(r + c - (rows - 1)) <= band) {

                        gl.alive = gl.next = true;
                        gl.color = RED_MERCURY;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GOLD;
                    }
                    break;
                }
                case 6: { // Circle
                    float centerX = cols / 2.0f;
                    float centerY = rows / 2.0f;
                    int gap = 150;
                    int gapSq = gap * gap;
                    float radiusSquared = (rows / 3.0f) * (rows / 3.0f);
                    float radiusSquared2 = (rows / 6.0f) * (rows / 6.0f);
                    float radiusSquared3 = (rows / 9.0f) * (rows / 9.0f);
                    float dx = c - centerX;
                    float dy = r - centerY;
                    float dist = dx * dx + dy * dy;
                    if (dist <= radiusSquared && dist > radiusSquared2 + gapSq) {
                        gl.alive = gl.next = true;
                        gl.color = BLUE_PLANET;
                    } else if (dist > radiusSquared2 && dist < radiusSquared3 + gapSq ){
                        gl.alive = gl.next = true;
                        gl.color = SUN_YELLOW;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = URANUS_BLUE;
                    }
                    break;
                    }
                case 7: {
                    float cx = cols / 2.0f;
                    float cy = rows / 2.0f;
                
                    float dx = c - cx;
                    float dy = r - cy;
                    float dist = sqrtf(dx * dx + dy * dy);
                    float angle = atan2f(dy, dx);  // [-π, π]
                    angle = angle < 0 ? angle + 2.0f * M_PI : angle;
                
                    // Parameters
                    // float spacing = 6.0f;        // radial spacing per full rotation (~coil tightness)
                    // float thickness = 2.5f;      // thickness of the spiral band
                
                    // Spiral formula: r = spacing * theta
                    float r_cw = angle * spacing;
                    float diff_cw = fabs(dist - r_cw);
                
                    // Opposite spiral
                    float r_ccw = (2.0f * M_PI - angle) * spacing;
                    float diff_ccw = fabs(dist - r_ccw);
                
                    if (diff_cw < thickness || diff_ccw < thickness) {
                        gl.alive = gl.next = true;
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = PINK;
                    }
                    break;
                }
                case 8: { // border
                    int border = 50;  // thickness of border
                    if (r < border || r >= rows - border || c < border || c >= cols - border) {
                        gl.alive = gl.next = true;
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GOLD;
                    }
                    break;
                }
                case 9: {  // double border
                    int outer = 1;  // outer thickness
                    int inner = 50;  // inner offset
                    bool isOuter = (r < outer || r >= rows - outer || c < outer || c >= cols - outer);
                    bool isInner = (r >= inner && r < rows - inner && c >= inner && c < cols - inner);
                    if (isOuter || isInner) {
                        gl.alive = gl.next = true;
                        gl.color = NEPTUNE_PURPLE;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = SUN_YELLOW;
                    }
                    break;
                }
                case 10: { // concentric Rings
                    float cx = cols / 2.0f;
                    float cy = rows / 2.0f;
                    float dx = c - cx;
                    float dy = r - cy;
                    float dist = sqrtf(dx * dx + dy * dy);
                
                    //* ringSpacing :controls distance between rings
                    //* thickness : ring band thickness
                
                    float modVal = fmodf(dist, ringSpacing);
                    if (modVal < thickness) {
                        gl.alive = gl.next = true;
                        gl.color = GREEN;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = NEPTUNE_PURPLE;
                    }
                    break;
                }
                case 11: { // Radial beam
                    float cx = cols / 2.0f;
                    float cy = rows / 2.0f;
                    float dx = c - cx;
                    float dy = r - cy;
                
                    float angle = atan2f(dy, dx);  // range: [-π, π]
                    angle = angle < 0 ? angle + 2.0f * M_PI : angle;  // normalize to [0, 2π]
                
                    int numBeams = 16;         // number of sun rays
                    float beamWidth = M_PI * 2.0f / numBeams;  // angle between beams
                
                    int beamIndex = (int)(angle / beamWidth);
                    if (beamIndex % 2 == 0) {
                        gl.alive = gl.next = true;
                        gl.color = SUN_YELLOW;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = GREEN;
                    }
                    break;
                }
                case 12: {  // Animated Rotating Sunbeam
                    float cx = cols / 2.0f;
                    float cy = rows / 2.0f;
                    float dx = c - cx;
                    float dy = r - cy;
                
                    float angle = atan2f(dy, dx);
                    angle = angle < 0 ? angle + 2.0f * M_PI : angle;
                
                    int numBeams = 16;
                    float beamWidth = 2.0f * M_PI / numBeams;
                
                    float angularOffset = fmodf(framesCount * dt * 0.5f, 2.0f * M_PI);  // rotate over time
                    angle += angularOffset;
                
                    int beamIndex = (int)(angle / beamWidth);
                    if (beamIndex % 2 == 0) {
                        gl.alive = gl.next = true;
                        gl.color = SUN_YELLOW;
                    } else {
                        gl.alive = gl.next = false;
                        gl.color = URANUS_BLUE;
                    }
                    break;
                }
            case 13: {
                // int blockSize = 6;      // size of each square block
                // int band = 1;           // diagonal thickness
            
                int blockRow = r / blockSize;
                int blockCol = c / blockSize;
            
                int localR = r % blockSize;
                int localC = c % blockSize;
            
                // Diagonal type: choose one or alternate
                bool useForwardSlash = true;  // true = '/', false = '\'
            
                // Optional: alternate slashes like a checker
                // if ((blockRow + blockCol) % 2 == 0) useForwardSlash = true;
                // else useForwardSlash = false;
            
                bool isDiagonal = false;
            
                if (useForwardSlash) {
                    isDiagonal = abs(localR + localC - (blockSize - 1)) <= band;
                } else {
                    isDiagonal = abs(localR - localC) <= band;
                }
            
                if (isDiagonal) {
                    gl.alive = gl.next = true;
                    gl.color = ORANGE;
                } else {
                    gl.alive = gl.next = false;
                    gl.color = TAN;
                }
                break;
            }
        case 14: {
            // int blockSize = 6;  // size of each square
            // int border = 1;     // thickness of grid lines
            // int diagonalBand = 1;  // diagonal thickness
        
            int blockRow = r / blockSize;
            int blockCol = c / blockSize;
        
            int localR = r % blockSize;
            int localC = c % blockSize;
        
            bool isBorder = (localR < border || localR >= blockSize - border ||
                                localC < border || localC >= blockSize - border);
        
            // Diagonal type: '/' or '\' or alternating
            bool useForwardSlash = ((blockRow + blockCol) % 2 == 0);  // alternate per tile
        
            bool isDiagonal = false;
            if (useForwardSlash) {
                isDiagonal = abs(localR + localC - (blockSize - 1)) <= diagonalBand;
            } else {
                isDiagonal = abs(localR - localC) <= diagonalBand;
            }
        
            if (isBorder || isDiagonal) {
                gl.alive = gl.next = true;
                gl.color = make_uchar4(255, 200, 50, 255);  // gold-orange
            } else {
                gl.alive = gl.next = false;
                gl.color = make_uchar4(20, 20, 20, 255);  // dark background
            }
            break;
        }
        // case 15: { // rule 30
        //     int center = cols / 2;
        //     if (r == 0) {
        //         // First row: set single center cell alive
        //         gl.alive = gl.next = (c == center);
        //     } else {
        //         // Read previous row
        //         int rowAbove = r - 1;
        //         int idxLeft = rowAbove * cols + c - 1;
        //         int idxCenter = rowAbove * cols + c;
        //         int idxRight = rowAbove * cols + c + 1;
        
        //         bool left = (c > 0)              ? gamelife[idxLeft].alive   : false;
        //         bool mid  =                        gamelife[idxCenter].alive;
        //         bool right= (c < cols - 1)       ? gamelife[idxRight].alive  : false;
        
        //         // Apply Rule 30 logic:  new = left XOR (mid OR right)
        //         gl.alive = gl.next = left ^ (mid || right);
        //     }
        
        //     gl.color = gl.alive ? make_uchar4(255, 255, 255, 255) : make_uchar4(0, 0, 0, 255);
        //     break;
        // }
            case 15: {
                int center = cols / 2;
            
                // Expose rule via ImGui slider (external to this loop)
                // extern uint8_t rule;
            
                if (r == 0) {
                    // Seed: single dot at center
                    gl.alive = gl.next = (c == center);
                } else {
                    int rowAbove = r - 1;
                    int idxLeft = rowAbove * cols + c - 1;
                    int idxMid  = rowAbove * cols + c;
                    int idxRight= rowAbove * cols + c + 1;
            
                    bool left  = (c > 0)         ? gamelife[idxLeft].alive : false;
                    bool mid   = gamelife[idxMid].alive;
                    bool right = (c < cols - 1)  ? gamelife[idxRight].alive : false;
            
                    int pattern = (left << 2) | (mid << 1) | right;
                    gl.alive = gl.next = ((rule >> pattern) & 1);
                }
            
                gl.color = gl.alive ? WHITE : RED_MERCURY;
                break;
            }
                default: 
                    break;

            }
            
            gamelife.push_back(gl);
        }
    }
}

hipSurfaceObject_t CUDAHandler::MapSurfaceResouse()
{
    //* Map the resource for CUDA
    hipArray_t array;
    // glFinish();
    hipGraphicsMapResources(1, &cudaResource, 0);
    hipGraphicsSubResourceGetMappedArray(&array, cudaResource, 0, 0);

    //* Create a CUDA surface object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    hipSurfaceObject_t surface = 0;
    hipCreateSurfaceObject(&surface, &resDesc);
    return surface;
}
