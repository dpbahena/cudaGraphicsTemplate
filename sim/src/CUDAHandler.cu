#include "CUDAHandler.h"
#include "cudaKernels.cuh"
#include "cuda_utils.h"




CUDAHandler* CUDAHandler::instance = nullptr;

CUDAHandler::CUDAHandler(int width, int height, GLuint textureID) :  width(width), height(height)
{
    hipGraphicsGLRegisterImage(&cudaResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    instance = this; // store global reference (to be used for mouse and imGui User Interface (UI) operations)
}

CUDAHandler::~CUDAHandler()
{
    hipGraphicsUnregisterResource(cudaResource);
}

void CUDAHandler::updateDraw(float dt)
{
    this->dt = dt;

    

    //* Map the resource for CUDA
    hipArray_t array;
    // glFinish();
    hipGraphicsMapResources(1, &cudaResource, 0);
    hipGraphicsSubResourceGetMappedArray(&array, cudaResource, 0, 0);

    //* Create a CUDA surface object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    hipSurfaceObject_t surface = 0;
    hipCreateSurfaceObject(&surface, &resDesc);

    // clear graphics
    int threads = 16; 
    dim3 clearBlock(threads, threads);
    dim3 clearGrid((width + clearBlock.x -1) / clearBlock.x, (height + clearBlock.y - 1) / clearBlock.y);
    clearSurface_kernel<<<clearGrid, clearBlock>>>(surface, width, height, BLUE_PLANET);

    drawCircle_kernel<<<1, 1>>>(surface, width, height, width/2, height/2, 200, SUN_YELLOW, 1, 4, zoom, panX, panY);

    drawGlowingCircle<<<1, 1>>>(surface, width, height, width / 2, height / 2, 50, RED_MERCURY, 1.5f, zoom, panX, panY);
    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());

    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &cudaResource);
}

