#include "CUDAHandler.h"
#include "cudaKernels.cuh"
#include "cuda_utils.h"




CUDAHandler* CUDAHandler::instance = nullptr;

CUDAHandler::CUDAHandler(int width, int height, GLuint textureID) :  width(width), height(height)
{
    hipGraphicsGLRegisterImage(&cudaResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    instance = this; // store global reference (to be used for mouse and imGui User Interface (UI) operations)
}

CUDAHandler::~CUDAHandler()
{
    hipGraphicsUnregisterResource(cudaResource);
}

void CUDAHandler::updateDraw(float dt)
{
    this->dt = dt;

    

    //* Map the resource for CUDA
    hipArray_t array;
    glFinish();
    hipGraphicsMapResources(1, &cudaResource, 0);
    hipGraphicsSubResourceGetMappedArray(&array, cudaResource, 0, 0);

    //* Create a CUDA surface object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;

    hipSurfaceObject_t surface = 0;
    hipCreateSurfaceObject(&surface, &resDesc);

    // hipError_t err = hipGraphicsMapResources(1, &cudaResource, 0);
    // std::cout << "MapResources error: " << hipGetErrorString(err) << std::endl;

    // hipError_t err = hipGraphicsMapResources(1, &cudaResource, 0);
    // if (err != hipSuccess) {
    //     std::cerr << "MapResources failed: " << hipGetErrorString(err) << std::endl;
    // }
    // hipGraphicsSubResourceGetMappedArray(&array, cudaResource, 0, 0);

    // printf("Width: %d, Height: %d\n", width, height);
    // clear graphics
    int threads = 16; 
    dim3 clearBlock(threads, threads);
    dim3 clearGrid((width + clearBlock.x -1) / clearBlock.x, (height + clearBlock.y - 1) / clearBlock.y);
    clearSurface_kernel<<<clearGrid, clearBlock>>>(surface, width, height, BLUE_PLANET);

    drawCircle_kernel<<<1, 1>>>(surface, width, height, width/2, height/2, 200, SUN_YELLOW, 1, 4 );
    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());

    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &cudaResource);
}

