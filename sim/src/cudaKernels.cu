#include "hip/hip_runtime.h"
#include "cudaKernels.cuh"
#include "hip/hip_vector_types.h"

__device__
void drawPixel(hipSurfaceObject_t surface, int x, int y, uchar4 color, int width, int height)
{
    if (x >= 0 && x < width && y >= 0 && y < height) {
        surf2Dwrite(color, surface, x * sizeof(uchar4), y);
    }
}

__device__ void drawLine(hipSurfaceObject_t surface, int x0, int y0, int x1, int y1, uchar4 color, int width, int height)
{
    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = -abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = dx + dy, e2;

    while (true) {
        drawPixel(surface, x0, y0, color, width, height);
        if (x0 == x1 && y0 == y1) break;
        e2 = 2 * err;
        if (e2 >= dy) { err += dy; x0 += sx; }
        if (e2 <= dx) { err += dx; y0 += sy; }
    }
}

__device__ void drawCircleOutline(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    const int segments = 36; // More segments = smoother circle
    for (int i = 0; i < segments; ++i) {
        float theta0 = (2.0f * M_PI * i) / segments;
        float theta1 = (2.0f * M_PI * (i + 1)) / segments;
        
        int x0 = cx + radius * cosf(theta0);
        int y0 = cy + radius * sinf(theta0);
        int x1 = cx + radius * cosf(theta1);
        int y1 = cy + radius * sinf(theta1);

        drawLine(surface, x0, y0, x1, y1, color, width, height);
    }
}

__device__ void drawRing(hipSurfaceObject_t surface, float cx, float cy, float radius, float thickness, uchar4 color, int width, int height)
{
    int minX = max(0, int(cx - radius - thickness));
    int maxX = min(width - 1, int(cx + radius + thickness));
    int minY = max(0, int(cy - radius - thickness));
    int maxY = min(height - 1, int(cy + radius + thickness));

    for (int y = minY; y <= maxY; ++y) {
        for (int x = minX; x <= maxX; ++x) {
            float dx = x - cx;
            float dy = y - cy;
            float dist = sqrtf(dx * dx + dy * dy);
            if (fabsf(dist - radius) < thickness) {
                surf2Dwrite(color, surface, x * sizeof(uchar4), y);
            }
        }
    }
}


__device__ void drawFilledCircle(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    int rSquared = radius * radius;
    for (int dy = -radius; dy <= radius; ++dy) {
        int y = cy + dy;
        if (y < 0 || y >= height) continue;

        for (int dx = -radius; dx <= radius; ++dx) {
            int x = cx + dx;
            if (x < 0 || x >= width) continue;

            if (dx * dx + dy * dy <= rSquared) {
                drawPixel(surface, x, y, color, width, height);
            }
        }
    }
}
__device__ uchar4 blend(uchar4 dest, uchar4 src) {
    float alpha = src.w / 255.0f;  // src alpha
    uchar4 result;
    result.x = (unsigned char) ((1.0f - alpha) * dest.x + alpha * src.x);
    result.y = (unsigned char) ((1.0f - alpha) * dest.y + alpha * src.y);
    result.z = (unsigned char) ((1.0f - alpha) * dest.z + alpha * src.z);

    result.w  = 255;  // keep opage or preserve dest.w if needed
    return result;
}
__device__ void drawBlendedFilledCircle(hipSurfaceObject_t surface, int cx, int cy, int radius, uchar4 color, int width, int height) {
    int rSquared = radius * radius;
    for (int dy = -radius; dy <= radius; ++dy) {
        int y = cy + dy;
        if (y < 0 || y >= height) continue;

        for (int dx = -radius; dx <= radius; ++dx) {
            int x = cx + dx;
            if (x < 0 || x >= width) continue;

            if (dx * dx + dy * dy <= rSquared) {
                uchar4 dst;
                surf2Dread(&dst, surface, x * sizeof(uchar4), y);
                uchar4 blended = blend(dst, color);

                drawPixel(surface, x, y, blended, width, height);
            }
        }
    }
}

__global__ void clearSurface_kernel(hipSurfaceObject_t surface, int width, int height, uchar4 color) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height) return;
    surf2Dwrite(color, surface, x * sizeof(uchar4), y);
}



__global__ void drawCircle_kernel(hipSurfaceObject_t surface, int width, int height, int cx, int cy, int radius, uchar4 color, bool outline, int thickness, float zoom, float panX, float panY)
{
    cx = (int) (cx + panX) * zoom + width / 2.0f;
    cy = (int) (cy + panY) * zoom + height/ 2.0f;

    radius *= zoom;
    if (outline){ 
        if (thickness > 0) 
            drawRing(surface, cx, cy, radius, thickness, color, width, height);
        else 
            drawCircleOutline(surface, cx, cy, radius, color, width, height);
    } else
        drawBlendedFilledCircle(surface, cx, cy, radius, color, width, height);
}


__global__ void drawGlowingCircle_kernel(hipSurfaceObject_t surface, int width, int height, int cx, int cy, int radius, uchar4 color, float glowExtent, float zoom, float panX, float panY) {

    cx = (int) (cx + panX) * zoom + width / 2.0f;
    cy = (int) (cy + panY) * zoom + height/ 2.0f;
    
    radius *= zoom;
    drawFilledCircle(surface, cx, cy, radius, color, width, height);

    int rSquared = radius * radius;
    float glowRadius = glowExtent * radius;
    float glowRadiusSquared = glowRadius * glowRadius;

    for (int dy = -glowRadius; dy <= glowRadius; ++dy) {
        int y = cy + dy;
        if (y < 0 || y >= height) continue;

        for (int dx = -glowRadius; dx <= glowRadius; ++dx) {
            int x = cx + dx;
            if (x < 0 || x >= width) continue;

            float distSquared = dx * dx + dy * dy;
            
            if (distSquared > rSquared && distSquared <= glowRadiusSquared) { // Only outside solid
                float intensity = 1.0f - (sqrtf(distSquared) - radius) / (glowRadius - radius);
                intensity = fmaxf(intensity, 0.0f);
                intensity = fminf(intensity, 1.0f);

                uchar4 outColor = make_uchar4(
                    min(255, (int)(color.x * intensity)),
                    min(255, (int)(color.y * intensity)),
                    min(255, (int)(color.z * intensity)),
                    color.w
                );

                // --- Blending with background ---
                uchar4 oldColor;
                surf2Dread(&oldColor, surface, x * sizeof(uchar4), y);

                uchar4 blendedColor = make_uchar4(
                    min(255, oldColor.x + outColor.x),
                    min(255, oldColor.y + outColor.y),
                    min(255, oldColor.z + outColor.z),
                    min(255, oldColor.w + outColor.w)
                );


                drawPixel(surface, x, y, blendedColor, width, height);
            }
        }
    }
}

__global__ void drawGlowingCircle_kernel(hipSurfaceObject_t surface, int width, int height, int cx, int cy, int radius, uchar4 color, float glowExtent, float xMin, float yMin, float zoom, float panX, float panY)
{
    int local_x = threadIdx.x + blockIdx.x * blockDim.x;
    int local_y = threadIdx.y + blockIdx.y * blockDim.y;

    int x = xMin + local_x;
    int y = yMin + local_y;


    if (x >= width || y >= height) return;

    // Convert screen pixel (x,y) to world space
    float worldX = (x - width * 0.5f) / zoom - panX;
    float worldY = (y - height * 0.5f) / zoom - panY;
    

    float dx = worldX - cx;
    float dy = worldY - cy;

    
    float distSquared = dx * dx + dy * dy;
    // int rSquared = radius * radius;
    // float glowRadius = glowExtent * radius;
    // float glowRadiusSquared = glowRadius * glowRadius;
    float falloffPower = 2.f;

    // Instead of sqrt(distSquared), compare distSquared directly (faster than fsqrt)
    float glowRadiusSquared = (glowExtent * radius) * (glowExtent * radius);

    if (distSquared <= glowRadiusSquared) {
        float normalizedSquared = distSquared / glowRadiusSquared;
        // * Option 1 - Linear distance fade
        // float intensity = 1.0f - normalizedSquared; 

        // * Option 2 - quadratic distance fade (optional softer/harder)

        /** falloffPower = 1.0f → normal fade
            falloffPower = 2.0f → steeper fade  (fast fade)
            falloffPower = 0.5f → softer fade 
        */
        float intensity = powf(1.0f - normalizedSquared, falloffPower);
    
        intensity = fmaxf(intensity, 0.0f);
        intensity = fminf(intensity, 1.0f);
    
        uchar4 newColor = make_uchar4(
            min(255, (int)(color.x * intensity)),
            min(255, (int)(color.y * intensity)),
            min(255, (int)(color.z * intensity)),
            (unsigned char)(color.w * intensity)
        );
    
        uchar4 oldColor;
        surf2Dread(&oldColor, surface, x * sizeof(uchar4), y);
    
        uchar4 blendedColor = make_uchar4(
            min(255, oldColor.x + newColor.x),
            min(255, oldColor.y + newColor.y),
            min(255, oldColor.z + newColor.z),
            min(255, oldColor.w + newColor.w)
        );

        
        // Fade color toward background with slight blending
        // float blendFactor = 0.5f; // between 0 (overwrite) and 1 (fully additive)

        // uchar4 oldColor;
        // surf2Dread(&oldColor, surface, x * sizeof(uchar4), y);

        // uchar4 blendedColor = make_uchar4(
        //     min(255, (unsigned char)(oldColor.x * (1.0f - blendFactor) + newColor.x * blendFactor)),
        //     min(255, (unsigned char)(oldColor.y * (1.0f - blendFactor) + newColor.y * blendFactor)),
        //     min(255, (unsigned char)(oldColor.z * (1.0f - blendFactor) + newColor.z * blendFactor)),
        //     min(255, (unsigned char)(oldColor.w * (1.0f - blendFactor) + newColor.w * blendFactor))
        // );
    
        surf2Dwrite(blendedColor, surface, x * sizeof(uchar4), y);
        // surf2Dwrite(newColor, surface, x * sizeof(uchar4), y);
    }
    
}


__global__ void drawRing_kernel(hipSurfaceObject_t surface, int width, int height, float centerX_world, float centerY_world, float radius, uchar4 color, float thickness, int xMin, int yMin, float zoom, float panX, float panY) 
{
    int local_x = threadIdx.x + blockIdx.x * blockDim.x;
    int local_y = threadIdx.y + blockIdx.y * blockDim.y;

    int x = xMin + local_x;
    int y = yMin + local_y;


    if (x >= width || y >= height) return;

    // Convert screen pixel (x,y) to world space coordinates
    float worldX = (x - width * 0.5f) / zoom - panX;
    float worldY = (y - height * 0.5f) / zoom - panY;
    // radius *= zoom;

    float dx = worldX - centerX_world;
    float dy = worldY - centerY_world;
    
    float dist = sqrtf(dx * dx + dy * dy);

    // Draw pixel if it falls within the ring (distance ± thickness)
    if (fabsf(dist - radius) < thickness) {
        surf2Dwrite(color, surface, x * sizeof(uchar4), y);
    }


}

/**
 * -- SHARED MEMORY -- in this case can reduce redundant calculations, especially if multiple pixels reuse the same values like zoom, panX, panY, or even values like centerX_world and centerY_world.
    But since shared memory is per block, the biggest gain would come from:
    Avoiding redundant math inside each block
    Optionally using it to cache small constant values shared across threads

    * Use shared memory to cache:

    centerX_world
    centerY_world
    radius
    thickness
    zoom, panX, panY
    These are constant across the whole block.
 */


 __global__ void drawRing_sharedMemory_kernel(hipSurfaceObject_t surface,
    int width, int height,
    float centerX_world, float centerY_world, float radius,
    uchar4 color, float thickness,
    int xMin, int yMin,
    float zoom, float panX, float panY) {

        // Declare shared memory (one copy per block)
        __shared__ float s_centerX_world;
        __shared__ float s_centerY_world;
        __shared__ float s_radius;
        __shared__ float s_thickness;
        __shared__ float s_zoom;
        __shared__ float s_panX;
        __shared__ float s_panY;

        // Initialized shared memory by the first thread in the block
        if (threadIdx.x == 0 and threadIdx.y == 0) {
            s_centerX_world = centerX_world;
            s_centerY_world = centerY_world;
            s_radius = radius;
            s_thickness = thickness;
            s_zoom = zoom;
            s_panX = panX;
            s_panY = panY;
        }
        // Wait until all threads in the block to finish loading shared memory
        __syncthreads();

        int local_x = threadIdx.x + blockIdx.x * blockDim.x;
        int local_y = threadIdx.y + blockIdx.y * blockDim.y;

        int x = xMin + local_x;
        int y = yMin + local_y;


        if (x >= width || y >= height) return;

        // Convert screen pixel (x,y) to world space coordinates
        float worldX = (x - width * 0.5f) / s_zoom - s_panX;
        float worldY = (y - height * 0.5f) / s_zoom - s_panY;
        
        // Compute distance from current pixel to world center
        float dx = worldX - s_centerX_world;
        float dy = worldY - s_centerY_world;
        float dist = sqrtf(dx * dx + dy * dy);


        // Draw pixel if it falls within the ring (distance ± thickness)
        if (fabsf(dist - s_radius) < s_thickness) {
            surf2Dwrite(color, surface, x * sizeof(uchar4), y);
        }



    }
